#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "proto.h"
#include "gpumem.h"
#define NUM_STREAMS 4
#define CUDA_STREAM
//#define CUDA_DEFAULT_STREAM

#ifdef CUDA_STREAM
const int num_streams = NUM_STREAMS;
hipStream_t streams[num_streams];
#endif


__device__ int hash(char x, char y){
	int hashedIdx =	((x + y)*(x + y + 1)/2) + y;
	return hashedIdx;  
}


__global__  void calc_best_score(char* d_main_seq, char* d_sec_seq,int* d_hash_con,int* d_hash_semi_con, weight_t* weight, int offset,float* results,int res_block_len) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid > res_block_len)
    	return;
	for(int hyphen = 1; hyphen <= res_block_len; hyphen++){
		int delta = tid <= hyphen ? 0 : -1;
		if(tid == hyphen){
			results[tid + (hyphen-1)*res_block_len] = -weight->w4;
		}else if(d_main_seq[tid + offset] == d_sec_seq[tid + delta]){
			results[tid + (hyphen-1)*res_block_len] = weight->w1;
		}else if(d_hash_con[hash(d_main_seq[tid + offset], d_sec_seq[tid + delta])]){
			results[tid + (hyphen-1)*res_block_len] = -weight->w2;
		}else if(d_hash_semi_con[hash(d_main_seq[tid + offset],d_sec_seq[tid + delta])]){
			results[tid + (hyphen-1)*res_block_len] = -weight->w3;
		}else{
			results[tid + (hyphen-1)*res_block_len] = -weight->w4;
		}
	}
	
}

void hipError_t(hipError_t error, const char* message){
	
	if (error != hipSuccess)
    {
        fprintf(stderr, message, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

float* compute_on_gpu(char *d_main_seq, char *d_sec_seq,int* d_hash_con,int* d_hash_semi_con ,weight_t* weight, int offset,int res_len,int thread_id){
	
	// Allocate the result in the gpu's memory
	float *d_result = NULL;
	size_t resultArraySize = sizeof(float) * (res_len * res_len);
	float* res = (float*)malloc(resultArraySize);
	hipMalloc((void**) &d_result, resultArraySize);
	
	// Set the number of thread in each block
	int threadsPerBlock = 256;

	// Compute the grid size 
	int blocksPerGrid = (res_len*res_len + threadsPerBlock - 1) / threadsPerBlock;
	
	#ifdef CUDA_STREAM
    hipStreamCreate(&streams[thread_id]);
	calc_best_score<<<blocksPerGrid, threadsPerBlock,0,streams[thread_id]>>>(d_main_seq, d_sec_seq, d_hash_con, d_hash_semi_con,weight,offset, d_result,res_len);
	hipMemcpyAsync(res , d_result , res_len*res_len*sizeof(float), hipMemcpyDeviceToHost,streams[thread_id]);

	hipStreamDestroy(streams[thread_id]);
	#endif

	#ifdef CUDA_DEFAULT_STREAM
	calc_best_score<<<blocksPerGrid, threadsPerBlock>>>(d_main_seq, d_sec_seq, d_hash_con, d_hash_semi_con,weight,offset, d_result,res_len);
	hipMemcpy(res , d_result , res_len*res_len*sizeof(float), hipMemcpyDeviceToHost);
	#endif
	
	// Free the current result array from the device 
	hipError_t(hipFree(d_result),"Failed free result device");
	return res;
			
}



char* allocate_main_sequence_on_gpu(char* h_mainSequence){
	char *d_mainSequence = NULL;
	size_t mainSequenceSize = (strlen(h_mainSequence)) * sizeof(char);
	hipError_t(hipMalloc((void**) &d_mainSequence, mainSequenceSize),"Could not allocate main sequence");
    hipMemcpy(d_mainSequence, h_mainSequence, mainSequenceSize, hipMemcpyHostToDevice);
	return d_mainSequence;
}

char* allocate_sec_sequence_on_gpu(char* h_sec_seq){
	char *d_sec_seq = NULL;
	size_t sec_seq_len = (strlen(h_sec_seq)) * sizeof(char);
    hipError_t(hipMalloc((void**) &d_sec_seq, sec_seq_len),"Could allocate sec sequence");
	hipMemcpy(d_sec_seq, h_sec_seq, sec_seq_len, hipMemcpyHostToDevice);
   	
	return d_sec_seq;
}

int* allocate_hash_conservative_on_gpu(int* hashedCon){
	int* d_hashedCon = NULL;
	size_t hashingSize = (1 << 14) * sizeof(int);
	hipError_t(hipMalloc((void**) &d_hashedCon, hashingSize),"Could not allocate con-hash");
    hipMemcpy(d_hashedCon, hashedCon, hashingSize, hipMemcpyHostToDevice);
    return d_hashedCon;
}

int* allocate_hash_semi_conservative_on_gpu(int* hashedSemiCon){
	int* d_hashedSemiCon = NULL;
	size_t hashingSize = (1 << 14) * sizeof(int);
	hipError_t(hipMalloc((void**) &d_hashedSemiCon, hashingSize),"Could not allocate semi-hash");
    hipMemcpy(d_hashedSemiCon, hashedSemiCon, hashingSize, hipMemcpyHostToDevice);
	return d_hashedSemiCon;
}

weight_t* allocate_weight_on_gpu(weight_t* weight){
	weight_t* d_weight = NULL;
	size_t weight_size = sizeof(weight_t);
	hipError_t(hipMalloc((void**) &d_weight, weight_size),"Could not allocate weight");
    hipMemcpy(d_weight, weight, weight_size, hipMemcpyHostToDevice);
	return d_weight;
}

void free_shared_resources(char* d_mainSequence, int* d_hashedCon, int* d_hashedSemiCon,weight_t* d_weight){
	hipError_t(hipFree(d_hashedCon),"Failed to free hashed from the device");
	hipError_t(hipFree(d_hashedSemiCon),"Failed to free semi-hashed from the device");
	hipError_t(hipFree(d_mainSequence),"Failed to free main sequence from the device");
	hipError_t(hipFree(d_weight),"Failed to free weight from the device");
}

void free_sec_sequence(char* d_sec_seq){
	hipError_t(hipFree(d_sec_seq),"Failed free secondary sequence device");
}

